#include "hip/hip_runtime.h"
#include "cuda_accelerated_deformation_detection.h"

#include <opencv2/cudev/common.hpp>
#include <stdio.h>

#include "cuda_math_vectors_helper.cuh"

namespace garment_augmentation {
namespace cuda_optimization {

	__device__ float3 VectorRotationZ(float3 from, float3 to) {
		float3 crossed = Crossed(from, to);
		float dot = Dot(from,to);
		float crossed_norm = Norm(crossed);

		float3 third_base = (to - from*dot);
		Normalize(third_base);

		// local rotation matrix
		float lrm_a = dot;
		float lrm_b = -crossed_norm;
		float lrm_c = 0.f;
		float lrm_d = crossed_norm;
		float lrm_e = dot;
		float lrm_f = 0.f;
		float lrm_g = 0.f;
		float lrm_h = 0.f;
		float lrm_i = 1.f;

		// to local space transformation matrix
		float lsm_a = from.x;
		float lsm_b = third_base.x;
		float lsm_c = -crossed.x;
		float lsm_d = from.y;
		float lsm_e = third_base.y,
		float lsm_f = -crossed.y;
		float lsm_g = from.z;
		float lsm_h = third_base.z,
		float lsm_i = -crossed.z;

		float determinant_local_space_matrix = 
			lsm_a * lsm_e * lsm_i
			+ lsm_b * lsm_f * lsm_g
			+ lsm_d * lsm_h * lsm_c
			- lsm_g * lsm_e * lsm_c
			- lsm_d * lsm_b * lsm_i
			- lsm_h * lsm_f * lsm_a;

		// to world space transformation matrix
		float wsm_a = (lsm_e * lsm_i - lsm_h * lsm_f) / determinant_local_space_matrix;
		float wsm_b = (-lsm_b * lsm_i + lsm_h * lsm_c) / determinant_local_space_matrix;
		float wsm_c = (lsm_b * lsm_f - lsm_e * lsm_c) / determinant_local_space_matrix;
		float wsm_d = (-lsm_d * lsm_i + lsm_g * lsm_f) / determinant_local_space_matrix;
		float wsm_e = (lsm_a * lsm_i - lsm_g * lsm_c) / determinant_local_space_matrix;
		float wsm_f = (-lsm_a * lsm_f + lsm_d * lsm_c) / determinant_local_space_matrix;
		float wsm_g = (lsm_d * lsm_h - lsm_g * lsm_e) / determinant_local_space_matrix;
		float wsm_h = (-lsm_a * lsm_h + lsm_g * lsm_b) / determinant_local_space_matrix;
		float wsm_i = (lsm_a * lsm_e - lsm_d * lsm_b) / determinant_local_space_matrix;

		float3 transform_on_z;
		transform_on_z.x =
			lsm_g*(lrm_a*wsm_a + lrm_b*wsm_d + lrm_c*wsm_g) + 
			lsm_h*(lrm_d*wsm_a + lrm_e*wsm_d + lrm_f*wsm_g) + 
			lsm_i*(lrm_g*wsm_a + lrm_h*wsm_d + lrm_i*wsm_g);
		transform_on_z.y =
			lsm_g*(lrm_a*wsm_b + lrm_b*wsm_e + lrm_c*wsm_h) +
			lsm_h*(lrm_d*wsm_b + lrm_e*wsm_e + lrm_f*wsm_h) +
			lsm_i*(lrm_g*wsm_b + lrm_h*wsm_e + lrm_i*wsm_h);
		transform_on_z.z =
			lsm_g*(lrm_a*wsm_c + lrm_b*wsm_f + lrm_c*wsm_i) +
			lsm_h*(lrm_d*wsm_c + lrm_e*wsm_f + lrm_f*wsm_i) +
			lsm_i*(lrm_g*wsm_c + lrm_h*wsm_f + lrm_i*wsm_i);

		return transform_on_z;
	}

	__device__ float DeltaDepth(const PtrStepSz<float3> world_coordinates, const float3 average_center, const float3 transformation, int x, int y, int half_width) {
		float delta_depth = 0;
		int nb_points = 0;

		for (int row = y - half_width; row <= y + half_width; row++) {
			for (int col = x - half_width; col <= x + half_width; col++) {
				if (!isinf(world_coordinates(row, col).z)) {
					float3 point = world_coordinates(row, col) - average_center; // Relative to the middle of the patch
					delta_depth += point.x * transformation.x + point.y * transformation.y + point.z * transformation.z; // Rotation to the normal of the patch
					nb_points++;
				}
			}
		}

		return delta_depth / nb_points;
	}

	// detected_deformations contain 0 if the deformation value of the area < threshold or the size of the area if >= threshold
	// half_window_width should be equal to floor(window_width/2)
	__global__ void ComputeDeformations(const PtrStepSz<float3> world_coordinates, const float threshold,
		const int half_window_width, PtrStepSz<uchar> detected_deformations) {
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if (x < world_coordinates.cols - half_window_width && y < world_coordinates.rows - half_window_width && x >= half_window_width && y >= half_window_width)
		{
			if (isinf(world_coordinates(x,y).z)) {
				detected_deformations(y, x) = 0;
				return;
			}
			float3 top_left_pt = world_coordinates(y - half_window_width, x - half_window_width);
			if (isinf(top_left_pt.z)) {
				detected_deformations(y, x) = 0;
				return;
			}
			float3 top_right_pt = world_coordinates(y - half_window_width, x + half_window_width);
			if (isinf(top_right_pt.z)) {
				detected_deformations(y, x) = 0;
				return;
			}
			float3 bottom_left_pt = world_coordinates(y + half_window_width, x - half_window_width);
			if (isinf(bottom_left_pt.z)) {
				detected_deformations(y, x) = 0;
				return;
			}
			float3 bottom_right_pt = world_coordinates(y + half_window_width, x + half_window_width);
			if (isinf(bottom_right_pt.z)) {
				detected_deformations(y, x) = 0;
				return;
			}

			float3 average_center = (top_left_pt + top_right_pt + bottom_left_pt + bottom_right_pt) / 4;

			float3 local_z_axis = Crossed(top_right_pt - top_left_pt, top_left_pt - bottom_left_pt) * -1;
			Normalize(local_z_axis);
			float3 world_axis = make_float3(0, 0, 1);
			float3 to_local_z = VectorRotationZ(local_z_axis,world_axis);

			float deformation = DeltaDepth(world_coordinates, average_center, to_local_z, x, y, half_window_width);
			if (deformation > threshold) {
				detected_deformations(y, x) = half_window_width;
			}
			else {
				detected_deformations(y, x) = 0;
			}
		}
		else {
			detected_deformations(y, x) = 0;
		}
	}

	__host__ void AcceleratedDeformationsDetection(const GpuMat world_coordinates, const float threshold,
	const int window_width, GpuMat detected_deformations, cv::cuda::Stream& s) {
		hipStream_t stream = cv::cuda::StreamAccessor::getStream(s);

		int half_window_width;
		if (window_width % 2 == 0)
			half_window_width = window_width / 2;
		else
			half_window_width = (window_width - 1) / 2;

		dim3 block(32, 8);
		dim3 grid((world_coordinates.cols + block.x - 1) / block.x, (world_coordinates.rows + block.y - 1) / block.y);
		ComputeDeformations <<<grid, block, 0, stream>>>(world_coordinates,threshold,half_window_width,detected_deformations);

		if (stream == 0)
			hipDeviceSynchronize();
	}

}
}