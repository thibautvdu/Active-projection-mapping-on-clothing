#include "hip/hip_runtime.h"
#include "deformation_detection.h"

#include <opencv2/cudev/common.hpp>
#include <stdio.h>

using namespace cv::cuda;

namespace garment_augmentation {
namespace cuda_optimization {

	const float k_float_zero = 0.0001; // in meters


	// Vector helpers
	__device__ float3 operator/(const float3 &a, float scal) {
		return make_float3(a.x / scal, a.y / scal, a.z / scal);
	}

	__device__ float3 operator*(const float3 &a, float scal) {
		return make_float3(a.x * scal, a.y * scal, a.z * scal);
	}

	__device__ float3 operator-(const float3 &a, const float3 &b) {
		return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
	}

	__device__ float3 Crossed(const float3 &a, const float3 &b) {
		return make_float3(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x);
	}

	__device__ float Dot(const float3 &a, const float3 &b) {
		return a.x*b.x + a.y*b.y + a.z*b.z;
	}

	__device__ float Norm(const float3 &a) {
		return a.x + a.y + a.z;
	}

	__device__ void Normalize(float3 &a) {
		float norm = Norm(a);
		a = a / norm;
	}

	__device__ float3 VectorRotationZ(float3 from, float3 to) {
		float3 crossed = Crossed(from, to);
		float dot = Dot(from,to);
		float crossed_norm = Norm(crossed);

		float3 third_base = (to - from*dot);
		Normalize(third_base);

		// to local space transformation matrix
		float lstm_a = from.x;
		float lstm_b = third_base.x;
		float lstm_c = -crossed.x;
		float lstm_d = from.y;
		float lstm_e = third_base.y,
		float lstm_f = -crossed.y;
		float lstm_g = from.z;
		float lstm_h = third_base.z,
		float lstm_i = -crossed.z;

		float determinant_local_space_matrix = 
			lstm_a * lstm_e * lstm_i
			+ lstm_b * lstm_f * lstm_g
			+ lstm_d * lstm_h * lstm_c
			- lstm_g * lstm_e * lstm_c
			- lstm_d * lstm_b * lstm_i
			- lstm_h * lstm_f * lstm_a;

		float local_space_invert_a = (lstm_e * lstm_i - lstm_h * lstm_f) / determinant_local_space_matrix;
		float local_space_invert_b = (-lstm_b * lstm_i + lstm_h * lstm_c) / determinant_local_space_matrix;
		float local_space_invert_c = (lstm_b * lstm_f - lstm_e * lstm_c) / determinant_local_space_matrix;
		float local_space_invert_d = (-lstm_d * lstm_i + lstm_g * lstm_f) / determinant_local_space_matrix;
		float local_space_invert_e = (lstm_a * lstm_i - lstm_g * lstm_c) / determinant_local_space_matrix;
		float local_space_invert_f = (-lstm_a * lstm_f + lstm_d * lstm_c) / determinant_local_space_matrix;
		float local_space_invert_g = (lstm_d * lstm_h - lstm_g * lstm_e) / determinant_local_space_matrix;
		float local_space_invert_h = (-lstm_a * lstm_h + lstm_g * lstm_b) / determinant_local_space_matrix;
		float local_space_invert_i = (lstm_a * lstm_e - lstm_d * lstm_b) / determinant_local_space_matrix;

		float3 transform_on_z;
		transform_on_z.x = 
			from.z*dot*local_space_invert_a + from.z*-crossed_norm*local_space_invert_c + from.z*0.f*local_space_invert_g +
			third_base.z*crossed_norm*local_space_invert_a + third_base.z*dot*local_space_invert_c + third_base.z*0.f*local_space_invert_g +
			-crossed.z*0.f*local_space_invert_a + -crossed.z*0.f*local_space_invert_c + -crossed.z*1.f*local_space_invert_g;
		transform_on_z.y =
			from.z*dot*local_space_invert_b + from.z*-crossed_norm*local_space_invert_e + from.z*0.f*local_space_invert_h +
			third_base.z*crossed_norm*local_space_invert_b + third_base.z*dot*local_space_invert_e + third_base.z*0.f*local_space_invert_h +
			-crossed.z*0.f*local_space_invert_b + -crossed.z*0.f*local_space_invert_e + -crossed.z*1.f*local_space_invert_h;
		transform_on_z.z =
			from.z*dot*local_space_invert_d + from.z*-crossed_norm*local_space_invert_f + from.z*0.f*local_space_invert_i +
			third_base.z*crossed_norm*local_space_invert_d + third_base.z*dot*local_space_invert_f + third_base.z*0.f*local_space_invert_i +
			-crossed.z*0.f*local_space_invert_d + -crossed.z*0.f*local_space_invert_f + -crossed.z*1.f*local_space_invert_i;

		return transform_on_z;
	}

	// detected_deformations contain 0 if the deformation value of the area < threshold or the size of the area if >= threshold
	// half_window_width should be equal to floor(window_width/2)
	__global__ void ComputeDeformations(const PtrStepSz<float3> world_coordinates, const float threshold,
		const int half_window_width, PtrStepSz<uchar> detected_deformations) {
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if (x < world_coordinates.cols - half_window_width && y < world_coordinates.rows - half_window_width && x >= half_window_width && y >= half_window_width)
		{
			float3 top_left_pt = world_coordinates(y - half_window_width, x - half_window_width);
			if (top_left_pt.z < k_float_zero) return;
			float3 top_right_pt = world_coordinates(y - half_window_width, x + half_window_width);
			if (top_right_pt.z < k_float_zero) return;
			float3 bottom_left_pt = world_coordinates(y + half_window_width, x - half_window_width);
			if (bottom_left_pt.z < k_float_zero) return;
			float3 bottom_right_pt = world_coordinates(y + half_window_width, x + half_window_width);
			if (bottom_right_pt.z < k_float_zero) return;
			
			float3 local_z_axis = Crossed(top_right_pt - top_left_pt, top_left_pt - bottom_left_pt);
			float3 world_axis = make_float3(0, 0, 0);
			
			float3 to_local_z = VectorRotationZ(world_axis, local_z_axis);

			float deformation = DeltaDepth(world_coordinates, to_local_z, x, y, half_window_width);
			if (deformation > threshold) {
				detected_deformations(y, x) = half_window_width;
			}
		}
	}

	__device__ float DeltaDepth(const PtrStepSz<float3> world_coordinates, const float3 transformation, int x, int y, int half_width) {
		float delta_depth;
		int nb_points = 0;

		for (int row = y - half_width; row < y + half_width; row++) {
			for (int col = y - half_width; col < x + half_width; col++) {
				if (world_coordinates(row,col).z > k_float_zero) {
					float3 point = world_coordinates(row, col) - world_coordinates(y, x); // Relative to the middle of the patch
					delta_depth += point.x * transformation.x + point.y * transformation.y + point.z * transformation.z; // Rotation to the normal of the patch
					nb_points++;
				}
			}
		}

		return delta_depth / nb_points;
	}

	__host__ void DetectDeformations(const PtrStepSz<float3> world_coordinates, const float threshold,
		const int half_window_width, hipStream_t stream, PtrStepSz<uchar> detected_deformations) {
		dim3 block(32, 8);
		dim3 grid((world_coordinates.cols + block.x - 1) / block.x, (world_coordinates.rows + block.y - 1) / block.y);
		ComputeDeformations <<<grid, block, 0, stream>>>(world_coordinates,threshold,half_window_width,detected_deformations);

		if (stream == 0)
			hipDeviceSynchronize();
	}

}
}